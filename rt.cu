#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_aabb.h>

//camera properties
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         fov, , );

//ray types
rtDeclareVariable(int, Phong, ,);
rtDeclareVariable(int, Shadow, ,);

//ray payloads
struct PerRayDataRadiance{
    float4 color;
};

struct PerRayDataShadow{
    int hit;
};

rtDeclareVariable(PerRayDataRadiance, rad_res, rtPayload, );
rtDeclareVariable(PerRayDataShadow, shadow_res, rtPayload, );

//material variables
rtTextureSampler<float4,2> tex0;
rtDeclareVariable(float4, diffuse, ,);
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(int, texCount, ,);

//geomerty buffers
rtBuffer<float3>vertex_buffer;
rtBuffer<float3>normal_buffer;
rtBuffer<int3>index_buffer;
rtBuffer<float2>texCoord_buffer;
rtDeclareVariable(int, hasTexCoord, , );

//intersection attributes
rtDeclareVariable(float2, texCoord, attribute texCoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(bool, useTex, attribute useTex, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

//ray and kernel size info
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

//output buffer
rtDeclareVariable(rtObject, top_object, , );
rtBuffer<float4,2> output0;

RT_PROGRAM void pinhole_camera(){
    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*V*fov + d.y*U*fov + W);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, Phong, 0.00000000001, RT_DEFAULT_MAX);
    PerRayDataRadiance rad_res;
    rad_res.color=make_float4(0.0f,0.0f,0.0f,0.0f);

	rtTrace(top_object, ray, rad_res);

	output0[launch_index] = rad_res.color;
	//output0[launch_index] = make_float4(1.f,0.f,0.f,0.f);
}

RT_PROGRAM void exception(){
    int code = rtGetExceptionCode();
    if(code==RT_EXCEPTION_STACK_OVERFLOW){
        output0[launch_index] = make_float4(1.f,0.f,0.f,0.f);
    }
}

RT_PROGRAM void closest_hit_radiance(){
    float4 color;

    float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);


    float3 lightPos=make_float3(0.f,-5.f,0.f);
    float3 pos=ray.origin+ray.direction*t_hit;
    float3 lightDir=normalize(lightPos-pos);

    float intensity=max(dot(lightDir,ffnormal),0.0f);

    if(texCount>0)
    {
        color=diffuse*intensity*tex2D(tex0,texCoord.x,texCoord.y);
    }
    else
    {
        color=diffuse*intensity;
    }
    PerRayDataShadow s;

    float dist=length(lightPos-pos);
    optix::Ray shadow = optix::make_Ray(pos,lightDir,Shadow,0.1, dist);
    s.hit=0;
    rtTrace(top_object,shadow,s);
    if(s.hit>0){
        color*=0.5;
    }
    rad_res.color=color;
}

RT_PROGRAM void any_hit_shadow(){
    shadow_res.hit=1;
    rtTerminateRay();
}

RT_PROGRAM void miss_radiance(){
    rad_res.color=make_float4(0.f,1.f,0.f,0.f);
}

RT_PROGRAM void miss_shadow(){
    shadow_res.hit=0;
}

RT_PROGRAM void intersectMesh(int primIdx){
    //get indices
    int3 id=index_buffer[primIdx];
    //get vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];
    //intersect ray with triangle
    float3 n;
    float t, beta, gamma;
    if(intersect_triangle(ray, v1, v2, v3, n, t, beta, gamma))
    {
        if(rtPotentialIntersection(t))
        {
            //loading normals
            float3 n1=normal_buffer[id.x];
            float3 n2=normal_buffer[id.y];
            float3 n3=normal_buffer[id.z];
            //loading texCoords
            if(hasTexCoord){
                float2 t1=texCoord_buffer[id.x];
                float2 t2=texCoord_buffer[id.y];
                float2 t3=texCoord_buffer[id.z];
                texCoord=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
                useTex=true;
            }
            else
            {
                texCoord=make_float2(1.0f,0.0f);
                useTex=false;
            }
            //setting attributes
            shading_normal=normalize((1.0f-beta-gamma)*n1 + beta*n2 +gamma*n3);
            geometric_normal=normalize(n);
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void boundingBoxMesh(int primIdx, float result[6]){
    //get indices
    int3 id=index_buffer[primIdx];
    //load vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];
    const float area = length(cross(v2-v1,v3-v1));
    optix::Aabb* aabb = (optix::Aabb*)result;
    if(area>0.0f)
    {
        aabb->m_min=fminf(fminf(v1,v2),v3);
        aabb->m_max=fmaxf(fmaxf(v1,v2),v3);
    }
    else
    {
        aabb->invalidate();
    }
}
