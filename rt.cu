#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_math.h>
#include <optixu/optixu_vector_types.h>
#include <optixu/optixu_aabb.h>

//light properties
rtDeclareVariable(float3, lightDir, , );

//camera properties
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float,         fov, , );

//ray types
rtDeclareVariable(int, Phong, ,);
rtDeclareVariable(int, Shadow, ,);

//ray payloads
struct PerRayDataRadiance{
    float4 color;
};

struct PerRayDataShadow{
    int hit;
};

rtDeclareVariable(PerRayDataRadiance, rad_res, rtPayload, );
rtDeclareVariable(PerRayDataShadow, shadow_res, rtPayload, );

//material variables
rtDeclareVariable(int, texCount, , );
rtTextureSampler<float4,2> tex0;
rtDeclareVariable(int, bumpCount, , );
rtTextureSampler<float,2> bump;
rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(float4, specular, , );
rtDeclareVariable(float, shininess, , );


//geomerty buffers
rtBuffer<float3>vertex_buffer;
rtBuffer<float3>normal_buffer;
rtBuffer<int3>index_buffer;
rtBuffer<float2>texCoord_buffer;
rtDeclareVariable(int, hasTexCoord, , );
rtBuffer<float3>tangent_buffer;
rtBuffer<float3>bitangent_buffer;

//intersection attributes
rtDeclareVariable(float2, texCoord, attribute texCoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float3, tangent, attribute tangent, );
rtDeclareVariable(float3, bitangent, attribute bitangent, );

//ray and kernel size info
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

//output buffer
rtDeclareVariable(rtObject, top_object, , );
rtBuffer<float4,2> output0;

RT_PROGRAM void pinhole_camera(){
    float ratio=float(launch_dim.x)/float(launch_dim.y);
    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*V*fov*ratio + d.y*U*fov + W);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, Phong, 0.00000000001, RT_DEFAULT_MAX);
    PerRayDataRadiance rad_res;
    rad_res.color=make_float4(0.0f,0.0f,0.0f,0.0f);

	rtTrace(top_object, ray, rad_res);

	output0[launch_index] = rad_res.color;
	//output0[launch_index] = make_float4(1.f,0.f,0.f,0.f);
}

RT_PROGRAM void exception(){
    int code = rtGetExceptionCode();
    if(code==RT_EXCEPTION_STACK_OVERFLOW){
        output0[launch_index] = make_float4(1.f,0.f,0.f,0.f);
    }
}

RT_PROGRAM void closest_hit_radiance(){
    float4 color;

    float3 local_normal=shading_normal;
    if(bumpCount){
        float delta_x=tex2D(bump,texCoord.x+0.001,texCoord.y)-tex2D(bump,texCoord.x-0.001,texCoord.y);
        float delta_y=tex2D(bump,texCoord.x,texCoord.y+0.001)-tex2D(bump,texCoord.x,texCoord.y-0.001);
        local_normal+=7.5f*(delta_x*tangent+delta_y*bitangent);
    }

    float3 world_geo_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal=normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, local_normal));
	float3 ffnormal=faceforward(world_shade_normal, -ray.direction, world_geo_normal);

    float3 pos=ray.origin+ray.direction*t_hit;

    float intensity=fmaxf(dot(ffnormal,-lightDir),0.f);

    if(texCount>0)
    {
        color=diffuse*tex2D(tex0,texCoord.x,texCoord.y);
    }
    else
    {
        color=diffuse;
    }

    if(intensity>0){
        optix::Ray shadow_ray =optix::make_Ray(pos,-lightDir,Shadow,0.1,RT_DEFAULT_MAX);
        PerRayDataShadow prds;
        rtTrace(top_object, shadow_ray, prds);
        if(prds.hit){
            intensity*=0.3f;
        }
    }
    color*=fmaxf(intensity,0.3f);
    rad_res.color=color;
}

RT_PROGRAM void any_hit_radiance(){
    float4 color;
    if(texCount>0)
    {
        color=diffuse*tex2D(tex0,texCoord.x,texCoord.y);
    }
    else
    {
        color=diffuse;
    }
    if(color.w==0.f) rtIgnoreIntersection();
}

RT_PROGRAM void any_hit_shadow(){
    float4 color;
    if(texCount>0)
    {
        color=diffuse*tex2D(tex0,texCoord.x,texCoord.y);
    }
    else
    {
        color=diffuse;
    }
    if(color.w==0.f) rtIgnoreIntersection();
    else{
        shadow_res.hit=1;
        rtTerminateRay();
    }
}

RT_PROGRAM void miss_radiance(){
    rad_res.color=make_float4(0.f,1.f,0.f,0.f);
}

RT_PROGRAM void miss_shadow(){
    shadow_res.hit=0;
}

RT_PROGRAM void intersectMesh(int primIdx){
    //get indices
    int3 id=index_buffer[primIdx];
    //get vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];
    //intersect ray with triangle
    float3 n;
    float t, beta, gamma;
    if(intersect_triangle(ray, v1, v2, v3, n, t, beta, gamma))
    {
        if(rtPotentialIntersection(t))
        {
            //loading normals
            float3 n1=normal_buffer[id.x];
            float3 n2=normal_buffer[id.y];
            float3 n3=normal_buffer[id.z];

            float3 t1=tangent_buffer[id.x];
            float3 t2=tangent_buffer[id.y];
            float3 t3=tangent_buffer[id.z];

            float3 b1=bitangent_buffer[id.x];
            float3 b2=bitangent_buffer[id.y];
            float3 b3=bitangent_buffer[id.z];

            //loading texCoords
            if(hasTexCoord){
                float2 t1=texCoord_buffer[id.x];
                float2 t2=texCoord_buffer[id.y];
                float2 t3=texCoord_buffer[id.z];
                texCoord=(1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3;
            }
            else
            {
                texCoord=make_float2(1.0f,0.0f);
            }
            //setting attributes
            shading_normal=normalize((1.0f-beta-gamma)*n1 + beta*n2 +gamma*n3);
            geometric_normal=normalize(n);
            tangent=normalize((1.0f-beta-gamma)*t1 + beta*t2 +gamma*t3);
            bitangent=normalize((1.0f-beta-gamma)*b1 + beta*b2 +gamma*b3);
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void boundingBoxMesh(int primIdx, float result[6]){
    //get indices
    int3 id=index_buffer[primIdx];
    //load vertices
    float3 v1=vertex_buffer[id.x];
    float3 v2=vertex_buffer[id.y];
    float3 v3=vertex_buffer[id.z];
    const float area = length(cross(v2-v1,v3-v1));
    optix::Aabb* aabb = (optix::Aabb*)result;
    if(area>0.0f)
    {
        aabb->m_min=fminf(fminf(v1,v2),v3);
        aabb->m_max=fmaxf(fmaxf(v1,v2),v3);
    }
    else
    {
        aabb->invalidate();
    }
}
